#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""


# define N 1024*1024
 

void fill_array(double *data) {
	for(int idx=0;idx<N;idx++)
		data[idx] = idx;
}


int verify_output(double *a, double *b, double *c) {
	int ok = 1;
	
	for (int idx=0;idx<N;idx++)
    	if (a[idx] + b[idx] != c[idx]) ok = 0;

	return ok;
}


__global__ void funcGPU(double *A, double *B, double *C){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
    C[index] = A[index] + B[index];
}


int main(){
    
    unsigned int blk_size[] = {1,2,4,8,32,64,128,192,256,384,512,1024};

	FILE *pFile;
	pFile = fopen("valores.txt", "w"); 

	double *A = (double*) malloc(N*sizeof(double));
	double *B = (double*) malloc(N*sizeof(double));
	double *C = (double*) malloc(N*sizeof(double));

	fill_array(A);
	fill_array(B);

	double *dA, *dB, *dC;

	hipMalloc(&dA, N*sizeof(double));
	hipMalloc(&dB, N*sizeof(double));
	hipMalloc(&dC, N*sizeof(double));

	
	for(int i=0; i<(sizeof blk_size/sizeof (unsigned int)); i++){
		unsigned int nBlocksX = N/blk_size[i];
		printf("Numero de Threads: %i\n", blk_size[i]);
		printf("Numero de Blocos: %i\n", nBlocksX);

		float averageKernel = 0;
		float averageTotal = 0;

		for (int j=1; j<=50; j++){
					
			float timeKernel, timeTotal;
			hipEvent_t startKernel, endKernel, startTotal, endTotal; 
			hipEventCreate(&startKernel);
			hipEventCreate(&endKernel);
			hipEventCreate(&startTotal);
			hipEventCreate(&endTotal);

			hipEventRecord(startTotal);
			hipMemcpy(dA, A, N*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(dB, B, N*sizeof(double), hipMemcpyHostToDevice);
	
			hipEventRecord(startKernel);
			funcGPU <<< {nBlocksX,1,1}, {blk_size[i],1,1} >>> (dA, dB, dC);
			hipEventRecord(endKernel);
			hipEventSynchronize(endKernel);
			hipEventElapsedTime(&timeKernel, startKernel, endKernel);
			
			hipMemcpy(C, dC, N*sizeof(double), hipMemcpyDeviceToHost);
			hipEventRecord(endTotal);
			hipEventSynchronize(endTotal);
			hipEventElapsedTime(&timeTotal, startTotal, endTotal);

			if (verify_output(A,B,C)){
				averageKernel = (averageKernel*(j-1)+timeKernel)/j;
				averageTotal = (averageTotal*(j-1)+timeTotal)/j;
			}
			
			else {
				j=j-1;
				printf("\n*** SOMA ERRADA! ***\n\n");
			}
		}
		
		float gopsTotal = (blk_size[i]*nBlocksX)/(averageTotal*1000000);
		float gopsKernel = (blk_size[i]*nBlocksX)/(averageKernel*1000000);
		
		printf("Tempo Total: %f ms\n", averageTotal);
		printf("Tempo Kernel: %f ms\n\n\n", averageKernel);
		fprintf(pFile, "%i;%f;%f\n", blk_size[i], gopsTotal, gopsKernel);

	}

	fclose(pFile);
	free(A);
	free(B);
	free(C);
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);


	pFile = fopen("modeloGPU.txt", "w"); 
	
	hipDeviceProp_t varProp;
	hipGetDeviceProperties(&varProp,0);
	fprintf(pFile, "%s", varProp.name);
	fclose(pFile);


	return 0;
}
